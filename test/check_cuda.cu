// filename: check_cuda.cu
#include <hip/hip_runtime.h>
#include <stdio.h>
 
__global__ void hello_from_gpu(void) {
    printf("Hello, World from GPU!\n");
}
 
int main() {
    int device_count;
    hipGetDeviceCount(&device_count);
 
    if (device_count == 0) {
        fprintf(stderr, "Did not detect any CUDA-capable devices.\n");
        return 1;
    }
 
    int device;
    for (device = 0; device < device_count; ++device) {
        hipDeviceProp_t prop;
        if (hipGetDeviceProperties(&prop, device) == hipSuccess) {
            printf("Device %d: %s\n", device, prop.name);
            hello_from_gpu<<<1, 1>>>();
            hipDeviceSynchronize();
        }
    }
 
    return 0;
}